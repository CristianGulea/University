#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string.h>
#include <math.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>

using namespace std;
using namespace cv;

void loadData(string& imagePath, uchar3* h_image, uchar3* d_image, uchar3* d_filterImage);

int rows, cols;
Mat rgbImage;

// Kernel function for red filter
__global__
void rgbToRedCudaKernel(uchar3* const rgbImage, uchar3* const redImage, int rows, int cols)
{
    const long pointIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (pointIndex < rows * cols) { // this is necessary only if too many threads are started
        uchar3 const imagePoint = rgbImage[pointIndex];
        redImage[pointIndex].x = (imagePoint.x- imagePoint.y) + (imagePoint.x- imagePoint.z);
        redImage[pointIndex].y = 0;
        redImage[pointIndex].z = 0;
    }
}

int main(int argc, char** argv)
{
    string imagePath = "4.jpg";
    string imageOutputPath = "4output.jpg";
    uchar3* h_image;
    uchar3* d_image;
    uchar3* d_filterImage;
    uchar3* h_filterImage;

    Mat inputImage;
    inputImage = imread(imagePath, IMREAD_COLOR);
    if (inputImage.empty()) 
    {
        cout << "Could not open or find the image" << endl;
        system("pause"); 
        exit(-1);
    }
    //convert BGR TO RGB
    cvtColor(inputImage, rgbImage, COLOR_BGR2RGB);
    cols = inputImage.cols;
    rows = inputImage.rows;
    int pixels = rows * cols;
    cout << cols << " " << rows << "\n";
    cout << pixels<<"\n";
    //copy rgb host image to device
    //alocate memory first
    h_image =(uchar3*) rgbImage.ptr<uchar3>(0);
    hipMalloc((void**)&(d_image), sizeof(uchar3) * pixels);
    hipMalloc((void**)&(d_filterImage), sizeof(uchar3) * pixels);
    hipMemset(d_filterImage, 0, sizeof(uchar3) * pixels);

    hipMemcpy(d_image, h_image, sizeof(uchar3) * pixels, hipMemcpyHostToDevice);

    const int blockThreadSize = 512;
    const int numberOfBlocks = 1 + ((rows * cols - 1) / blockThreadSize); // a/b rounded up
    const dim3 blockSize(blockThreadSize, 1, 1);
    const dim3 gridSize(numberOfBlocks, 1, 1);
    rgbToRedCudaKernel << <gridSize, blockSize  >> > (d_image, d_filterImage, rows,cols);
    hipDeviceSynchronize();

    size_t numPixels = rows * cols;
    h_filterImage =(uchar3*) malloc(sizeof(uchar3) * pixels);
    hipMemcpy(h_filterImage, d_filterImage, sizeof(uchar3) * numPixels, hipMemcpyDeviceToHost);

 
    Mat outputrgb(rows, cols, CV_8UC3,h_filterImage,0);
    Mat outputbgr(rows, cols, CV_8UC3);
    cvtColor(outputrgb, outputbgr, COLOR_RGB2BGR);
    cv::imshow("", outputbgr);
    waitKey(0);

    //output the image
    cv::imwrite(imageOutputPath, outputbgr);

    return 0;
}


